/*
 * HuberRegularizer_GPU_half.cu
 *
 *      Author: gac
 */


#include "HuberRegularizer_GPU_half.cuh"
#include "GPUConstant.cuh"
#include "HuberRegularizer_kernel_half.cuh"

HuberRegularizer_GPU_half::HuberRegularizer_GPU_half(double huberThreshold): huberThreshold(huberThreshold){}

HuberRegularizer_GPU_half::~HuberRegularizer_GPU_half(){}

double HuberRegularizer_GPU_half::getHuberThreshold() const
{
	return this->huberThreshold;
}

void HuberRegularizer_GPU_half::setHuberThreshold(double huberThreshold)
{
	this->huberThreshold = huberThreshold;
}

void HuberRegularizer_GPU_half::derivativeHuberFunction(Volume_GPU_half* volume, Volume_GPU_half* derivativeHuberVolume) const
{
	if(volume->isSameSize(derivativeHuberVolume))
	{
		CUDAArchitecture* gpuArch = ((Image3D_GPU<float>*)volume->getVolumeImage())->getCUDAArchitecture();

		dim3 dimBlock(gpuArch->getXThreadNb(), gpuArch->getYThreadNb(), gpuArch->getZThreadNb());
		dim3 dimGrid(gpuArch->getXBlockNb(), gpuArch->getYBlockNb(), gpuArch->getZBlockNb());

		unsigned long int xNb,yNb,zNb;
		xNb = volume->getXVolumePixelNb();
		yNb = volume->getYVolumePixelNb();
		zNb = volume->getZVolumePixelNb();
		float huberThres = this->getHuberThreshold();

		half* volumeData = volume->getVolumeData();
		half* derivativeHuberVolumeData = derivativeHuberVolume->getVolumeData();

		derivativeHuberFunction_k_half<<<dimGrid,dimBlock>>>((unsigned short*)volumeData, (unsigned short*)derivativeHuberVolumeData, huberThres, xNb*yNb*zNb);
		checkCudaErrors(hipDeviceSynchronize());
	}
	else
	{
		cout << "Volumes must have the same size" << endl;
		exit(EXIT_FAILURE);
	}
}

void HuberRegularizer_GPU_half::secondDerivativeHuberFunction(Volume_GPU_half* volume, Volume_GPU_half* secondDerivativeHuberVolume) const
{
	if(volume->isSameSize(secondDerivativeHuberVolume))
	{
		CUDAArchitecture* gpuArch = ((Image3D_GPU<float>*)volume->getVolumeImage())->getCUDAArchitecture();

		dim3 dimBlock(gpuArch->getXThreadNb(), gpuArch->getYThreadNb(), gpuArch->getZThreadNb());
		dim3 dimGrid(gpuArch->getXBlockNb(), gpuArch->getYBlockNb(), gpuArch->getZBlockNb());

		unsigned long int xNb,yNb,zNb;
		xNb = volume->getXVolumePixelNb();
		yNb = volume->getYVolumePixelNb();
		zNb = volume->getZVolumePixelNb();
		float huberThres = this->getHuberThreshold();

		half* volumeData = volume->getVolumeData();
		half* secondDerivativeHuberVolumeData = secondDerivativeHuberVolume->getVolumeData();

		secondDerivativeHuberFunction_k_half<<<dimGrid,dimBlock>>>((unsigned short*)volumeData, (unsigned short*)secondDerivativeHuberVolumeData, huberThres, xNb*yNb*zNb);
		checkCudaErrors(hipDeviceSynchronize());
	}
	else
	{
		cout << "Volumes must have the same size" << endl;
		exit(EXIT_FAILURE);
	}
}

void HuberRegularizer_GPU_half::getGradientRegularizationCriterion(Volume_GPU_half* volume, Volume_GPU_half* dJ, double* jReg, double* normdJProjReg, float lambda, int totalIterationIdx, int optimalStepIterationNb) const
{
	cout << "********** Start calcul of Huber regularization criterion jReg **********" << endl;

	float kernel_h[3] = {-1,0,1};
	float kernel_v[3] = {1,2,1};
	float kernel_p[3] = {1,2,1};

	CUDAArchitecture* gpuArch = ((Image3D_GPU<float>*)volume->getVolumeImage())->getCUDAArchitecture();

	Volume_GPU_half* gradientVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);

	Convolution3D_GPU_half convolver(kernel_h,kernel_v,kernel_p);
	convolver.doSeparableConvolution3D(volume,gradientVolume);

	*jReg=gradientVolume->getVolumeHuberNorm(this->getHuberThreshold());

	cout << "********** End calcul of Huber regularization criterion jReg **********" << endl;
	cout << "jReg = " << *jReg << endl;

	Volume_GPU_half* derivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	this->derivativeHuberFunction(gradientVolume, derivativeHuberVolume);

	Volume_GPU_half* djReg = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	convolver.doSeparableConvolution3D(derivativeHuberVolume,djReg);

	dJ->addVolume(djReg,lambda);

	delete djReg;

	delete derivativeHuberVolume;

	if(totalIterationIdx < optimalStepIterationNb)
	{

		Volume_GPU_half* secondDerivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
		this->secondDerivativeHuberFunction(gradientVolume, secondDerivativeHuberVolume);
		secondDerivativeHuberVolume->multVolume(dJ);
		convolver.doSeparableConvolution3D(secondDerivativeHuberVolume,gradientVolume);
		convolver.doSeparableConvolution3D(gradientVolume,secondDerivativeHuberVolume);
		secondDerivativeHuberVolume->multVolume(dJ);
		*normdJProjReg = secondDerivativeHuberVolume->getVolumeL1Norm<float>();

		delete secondDerivativeHuberVolume;
	}
	delete gradientVolume;
}

void HuberRegularizer_GPU_half::getGradientRegularizationCriterion(Volume_GPU_half* volume, Volume_GPU_half* dJ, Volume_GPU_half* p, Volume_GPU_half* olddJ, double* jReg, double* normdJProjReg, double* normdJ, double* normolddJ, double* beta, float lambda) const
{
	cout << "********** Start calcul of Huber regularization criterion jReg **********" << endl;

	float kernel_h[3] = {-1,0,1};
	float kernel_v[3] = {1,2,1};
	float kernel_p[3] = {1,2,1};

	CUDAArchitecture* gpuArch = ((Image3D_GPU<float>*)volume->getVolumeImage())->getCUDAArchitecture();

	Volume_GPU_half* gradientVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);

	Convolution3D_GPU_half convolver(kernel_h,kernel_v,kernel_p);
	convolver.doSeparableConvolution3D(volume,gradientVolume);

	*jReg=gradientVolume->getVolumeHuberNorm(this->getHuberThreshold());

	cout << "********** End calcul of Huber regularization criterion jReg **********" << endl;
	cout << "jReg = " << *jReg << endl;

	Volume_GPU_half* derivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	this->derivativeHuberFunction(gradientVolume, derivativeHuberVolume);

	Volume_GPU_half* djReg = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	convolver.doSeparableConvolution3D(derivativeHuberVolume,djReg);

	dJ->addVolume(djReg,lambda);

	delete djReg;

	delete derivativeHuberVolume;

	olddJ->diffVolume(dJ);
	*normdJ = dJ->scalarProductVolume<float>(olddJ);
	*beta = -1.0*(*normdJ)/(*normolddJ);
	*normdJ = dJ->getVolumeL2Norm<float>();
	*normolddJ = *normdJ;
	olddJ = dJ;
	cout << "Beta = " << *beta << endl;

	cout << "Start p Updating" << endl;
	p->diffVolume(*beta, dJ);
	cout << "End p Updating" << endl;

	Volume_GPU_half* secondDerivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	this->secondDerivativeHuberFunction(gradientVolume, secondDerivativeHuberVolume);
	secondDerivativeHuberVolume->multVolume(p);
	convolver.doSeparableConvolution3D(secondDerivativeHuberVolume,gradientVolume);
	convolver.doSeparableConvolution3D(gradientVolume,secondDerivativeHuberVolume);
	secondDerivativeHuberVolume->multVolume(p);
	*normdJProjReg = secondDerivativeHuberVolume->getVolumeL1Norm<float>();

	delete secondDerivativeHuberVolume;
	delete gradientVolume;
}

void HuberRegularizer_GPU_half::getLaplacianRegularizationCriterion(Volume_GPU_half* volume, Volume_GPU_half* dJ, double* jReg, double* normdJProjReg, float lambda, int totalIterationIdx, int optimalStepIterationNb) const
{
	cout << "********** Start calcul of Huber regularization criterion jReg **********" << endl;

	float kernel_h[3] = {-1,2,-1};
	float kernel_v[3] = {-1,2,-1};
	float kernel_p[3] = {-1,2,-1};

	CUDAArchitecture* gpuArch = ((Image3D_GPU<float>*)volume->getVolumeImage())->getCUDAArchitecture();

	Volume_GPU_half* gradientVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);

	Convolution3D_GPU_half convolver(kernel_h,kernel_v,kernel_p);
	convolver.doSeparableConvolution3D(volume,gradientVolume);

	//	gradientVolume->saveVolume("/espace/boulay/gradient2.v");

	*jReg=gradientVolume->getVolumeHuberNorm(huberThreshold);

	cout << "********** End calcul of Huber regularization criterion jReg **********" << endl;
	cout << "jReg = " << *jReg << endl;

	Volume_GPU_half* derivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	this->derivativeHuberFunction(gradientVolume, derivativeHuberVolume);

	Volume_GPU_half* djReg = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	convolver.doSeparableConvolution3D(derivativeHuberVolume,djReg);

	dJ->addVolume(djReg,lambda);

	delete djReg;

	delete derivativeHuberVolume;

	if(totalIterationIdx < optimalStepIterationNb)
	{

		Volume_GPU_half* secondDerivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);

		this->secondDerivativeHuberFunction(gradientVolume, secondDerivativeHuberVolume);
		secondDerivativeHuberVolume->multVolume(dJ);
		convolver.doSeparableConvolution3D(secondDerivativeHuberVolume,gradientVolume);
		convolver.doSeparableConvolution3D(gradientVolume,secondDerivativeHuberVolume);
		secondDerivativeHuberVolume->multVolume(dJ);
		*normdJProjReg = secondDerivativeHuberVolume->getVolumeL1Norm<float>();

		delete secondDerivativeHuberVolume;

	}
	delete gradientVolume;
}

void HuberRegularizer_GPU_half::getLaplacianRegularizationCriterion(Volume_GPU_half* volume, Volume_GPU_half* dJ, Volume_GPU_half* p, Volume_GPU_half* olddJ, double* jReg, double* normdJProjReg, double* normdJ, double* normolddJ, double* beta, float lambda) const
{
	cout << "********** Start calcul of Huber regularization criterion jReg **********" << endl;

	float kernel_h[3] = {-1,2,-1};
	float kernel_v[3] = {-1,2,-1};
	float kernel_p[3] = {-1,2,-1};

	CUDAArchitecture* gpuArch = ((Image3D_GPU<float>*)volume->getVolumeImage())->getCUDAArchitecture();

	Volume_GPU_half* gradientVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);

	Convolution3D_GPU_half convolver(kernel_h,kernel_v,kernel_p);
	convolver.doSeparableConvolution3D(volume,gradientVolume);

	*jReg=gradientVolume->getVolumeHuberNorm(huberThreshold);

	cout << "********** End calcul of Huber regularization criterion jReg **********" << endl;
	cout << "jReg = " << *jReg << endl;

	Volume_GPU_half* derivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	this->derivativeHuberFunction(gradientVolume, derivativeHuberVolume);

	Volume_GPU_half* djReg = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	convolver.doSeparableConvolution3D(derivativeHuberVolume,djReg);

	dJ->addVolume(djReg,lambda);

	delete djReg;

	delete derivativeHuberVolume;

	olddJ->diffVolume(dJ);
	*normdJ = dJ->scalarProductVolume<float>(olddJ);
	*beta = -1.0*(*normdJ)/(*normolddJ);
	*normdJ = dJ->getVolumeL2Norm<float>();
	*normolddJ = *normdJ;
	olddJ = dJ;
	cout << "Beta = " << *beta << endl;

	cout << "Start p Updating" << endl;
	p->diffVolume(*beta, dJ);
	cout << "End p Updating" << endl;

	Volume_GPU_half* secondDerivativeHuberVolume = new Volume_GPU_half(volume->getXVolumeSize(),volume->getYVolumeSize(),volume->getZVolumeSize(),volume->getXVolumePixelNb(),volume->getYVolumePixelNb(),volume->getZVolumePixelNb(),gpuArch);
	this->secondDerivativeHuberFunction(gradientVolume, secondDerivativeHuberVolume);
	secondDerivativeHuberVolume->multVolume(p);
	convolver.doSeparableConvolution3D(secondDerivativeHuberVolume,gradientVolume);
	convolver.doSeparableConvolution3D(gradientVolume,secondDerivativeHuberVolume);
	secondDerivativeHuberVolume->multVolume(p);
	*normdJProjReg = secondDerivativeHuberVolume->getVolumeL1Norm<float>();

	delete secondDerivativeHuberVolume;
	delete gradientVolume;
}

