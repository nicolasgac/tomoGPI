#include "hip/hip_runtime.h"
/*
 * BackProjector_half.cu
 *
 *      Author: gac
 */

#include "BackProjector.cuh"
#include "BackProjector_CPU.cuh"
#include "BackProjector_GPU.cuh"

#include "GPUConstant.cuh"

#include "backprojection3D_kernel_1reg_half.cuh"
#include "backprojection3D_kernel_2reg_half.cuh"
#include "backprojection3D_kernel_4reg_half.cuh"
#include "backprojection3D_kernel_8reg_half.cuh"
#include "backprojection3D_kernel_16reg_half.cuh"

#include "backprojection3D_kernel_1reg_half_UM.cuh"
#include "backprojection3D_kernel_2reg_half_UM.cuh"
#include "backprojection3D_kernel_4reg_half_UM.cuh"
#include "backprojection3D_kernel_8reg_half_UM.cuh"
#include "backprojection3D_kernel_16reg_half_UM.cuh"


/* BackProjector_half definition */
template<typename V, typename S>
BackProjector_half<V, S>::BackProjector_half(Acquisition* acquisition, Detector* detector,CUDABProjectionArchitecture* cudabackprojectionArchitecture, V* volume,char fdk) : acquisition(acquisition), detector(detector), cudabackprojectionArchitecture(cudabackprojectionArchitecture),volume(volume),fdk(fdk)
{
	double startAngle = acquisition->getStartAngle();
	unsigned short projectionNb = acquisition->getProjectionNb();
	float focusDetectorDistance = acquisition->getFocusDetectorDistance();
	float zVolumePixelSize = volume->getZVolumePixelSize();
	float vDetectorPixelSize = detector->getVDetectorPixelSize();

	this->alphaIOcylinderC = new float[projectionNb];
	this->betaIOcylinderC = new float[projectionNb];
	this->gammaIOcylinderC = vDetectorPixelSize/(focusDetectorDistance*zVolumePixelSize);

	double* phiValueTab = acquisition->getPhiValue();

	for (int p=0;p<projectionNb;p++)
	{
		alphaIOcylinderC[p] = cos(phiValueTab[p]);
		betaIOcylinderC[p] = sin(phiValueTab[p]);
	}


	alphaC = (M_PI*acquisition->getFocusObjectDistance()*acquisition->getFocusDetectorDistance())/(double)(projectionNb);
	betaC = (acquisition->getFocusDetectorDistance()/detector->getUDetectorPixelSize())*(acquisition->getFocusDetectorDistance()/detector->getUDetectorPixelSize());

}

template<typename V, typename S>
BackProjector_half<V, S>::~BackProjector_half()
{
	delete alphaIOcylinderC;
	delete betaIOcylinderC;
}

template<typename  V, typename S>
char BackProjector_half<V, S>::getFdk()
{
	return this->fdk;
}

template<typename  V, typename S>
void BackProjector_half<V, S>::setFdk(char fdk)
{
	this->fdk = fdk;
}
template<typename V, typename S>
unsigned long int BackProjector_half<V, S>::getProjectionNb()
{
	return this->getAcquisition()->getProjectionNb();
}

template<typename V, typename S>
Acquisition* BackProjector_half<V, S>::getAcquisition() const
{
	return this->acquisition;
}

template<typename V, typename S>
Detector* BackProjector_half<V, S>::getDetector() const
{
	return this->detector;
}


template<typename V, typename S>
CUDABProjectionArchitecture* BackProjector_half<V, S>::getCUDABProjectionArchitecture() const
{
	return this->cudabackprojectionArchitecture;
}


template<typename V, typename S>
V* BackProjector_half<V, S>::getVolume() const
{
	return this->volume;
}

template<typename V, typename S>
void BackProjector_half<V, S>::setAcquisition(Acquisition* acquisition)
{
	this->acquisition = acquisition;
}

template<typename V, typename S>
void BackProjector_half<V, S>::setDetector(Detector* detector)
{
	this->detector = detector;
}

template<typename V, typename S>
void BackProjector_half<V, S>::setCUDABProjectionArchitecture(CUDABProjectionArchitecture *cudabackprojectionArchitecture)
{
	this->cudabackprojectionArchitecture=cudabackprojectionArchitecture;
}

template<typename V, typename S>
void BackProjector_half<V, S>::setVolume(V* volume)
{
	this->volume = volume;
}


/* Copy backprojector constant */
template <typename V,typename S>
__host__ void BackProjector_half<V, S>::copyConstantGPU()
{
	unsigned long int projectionNb = (this->getAcquisition())->getProjectionNb();
	float xVolumeCenterPixel = this->getVolume()->getXVolumeCenterPixel();
	float yVolumeCenterPixel = this->getVolume()->getYVolumeCenterPixel();
	float zVolumeCenterPixel = this->getVolume()->getZVolumeCenterPixel();
	float xVolumePixelSize = this->getVolume()->getXVolumePixelSize();
	unsigned long int xVolumePixelNb = this->getVolume()->getXVolumePixelNb();
	unsigned long int yVolumePixelNb = this->getVolume()->getYVolumePixelNb();

	float fdd = this->getAcquisition()->getFocusDetectorDistance();
	float fod = this->getAcquisition()->getFocusObjectDistance();

	float uDetectorCenterPixel = this->getDetector()->getUDetectorCenterPixel();
	float vDetectorCenterPixel = this->getDetector()->getVDetectorCenterPixel();
	float uDetectorPixelSize = this->getDetector()->getUDetectorPixelSize();
	float vDetectorPixelSize = this->getDetector()->getVDetectorPixelSize();

	hipMemcpyToSymbol(HIP_SYMBOL(alphaIOcylinderC_GPU),this->alphaIOcylinderC,projectionNb*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(betaIOcylinderC_GPU),this->betaIOcylinderC,projectionNb*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(gammaIOcylinderC_GPU),&this->gammaIOcylinderC,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(xVolumeCenterPixel_GPU),&xVolumeCenterPixel,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(yVolumeCenterPixel_GPU),&yVolumeCenterPixel,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(zVolumeCenterPixel_GPU),&zVolumeCenterPixel,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(xVolumePixelSize_GPU),&xVolumePixelSize,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(xVolumePixelNb_GPU),&xVolumePixelNb,sizeof(unsigned long int));
	hipMemcpyToSymbol(HIP_SYMBOL(yVolumePixelNb_GPU),&yVolumePixelNb,sizeof(unsigned long int));

	hipMemcpyToSymbol(HIP_SYMBOL(focusDetectorDistance_GPU),&fdd,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(focusObjectDistance_GPU),&fod,sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(uDetectorCenterPixel_GPU),&uDetectorCenterPixel,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(vDetectorCenterPixel_GPU),&vDetectorCenterPixel,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(uDetectorPixelSize_GPU),&uDetectorPixelSize,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(vDetectorPixelSize_GPU),&vDetectorPixelSize,sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(alphaC_GPU),&this->alphaC,sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(betaC_GPU),&this->betaC,sizeof(float));
}



/* VIBackProjector_CPU definition */
VIBackProjector_CPU_half::VIBackProjector_CPU_half(Acquisition* acquisition, Detector* detector, CUDABProjectionArchitecture* cudabackprojectionArchitecture,Volume_CPU_half* volume,char fdk) : BackProjector_half<Volume_CPU_half,Sinogram3D_CPU_half>(acquisition, detector,cudabackprojectionArchitecture, volume, fdk){}

VIBackProjector_CPU_half::~VIBackProjector_CPU_half(){}

void VIBackProjector_CPU_half::doBackProjection(Volume_CPU_half *estimatedVolume,Sinogram3D_CPU_half* sinogram)
{
	std::cout << "VI BackProjection (half precision)" << std::endl;
	TGPUplan_retro_half<Volume_CPU_half, Sinogram3D_CPU_half>* plan;
	std::thread** threadID;
	int device;

	this->setVolume(estimatedVolume);

	unsigned int gpuNb = this->getCUDABProjectionArchitecture()->getComputingUnitNb();
	int nstreams=this->getCUDABProjectionArchitecture()->getBProjectionStreamsNb();
	printf("CUDA-capable device count: %d\n", gpuNb);

	struct hipDeviceProp_t prop_device;
	hipGetDeviceProperties(&prop_device,0);//propriétés du device 0
	int nb_bloc_phi=1;
	int nb_bloc_z_par_device=1;

	float fdd = this->getAcquisition()->getFocusDetectorDistance();
	float fod = this->getAcquisition()->getFocusObjectDistance();
	float vDetectorPixelSize = this->getDetector()->getVDetectorPixelSize();
	float vDetectorPixelNb = this->getDetector()->getVDetectorPixelNb();
	float vDetectorCenterPixel = this->getDetector()->getVDetectorCenterPixel();

	unsigned long int uSinogramPixelNb = sinogram->getUSinogramPixelNb();
	unsigned long int vSinogramPixelNb = sinogram->getVSinogramPixelNb();
	unsigned long int projectionSinogramNb = sinogram->getProjectionSinogramNb();

	unsigned long long int xThreadNb = this->getCUDABProjectionArchitecture()->getXThreadNb();
	unsigned long long int yThreadNb = this->getCUDABProjectionArchitecture()->getYThreadNb();
	unsigned long long int zThreadNb = this->getCUDABProjectionArchitecture()->getZThreadNb();

	float xVolumePixelSize = this->getVolume()->getXVolumePixelSize();
	float zVolumeCenterPixel = this->getVolume()->getZVolumeCenterPixel();
	unsigned long int xVolumePixelNb = this->getVolume()->getXVolumePixelNb();
	unsigned long int yVolumePixelNb = this->getVolume()->getYVolumePixelNb();
	unsigned long int zVolumePixelNb = this->getVolume()->getZVolumePixelNb();

	unsigned long long int xBlockNb = this->getCUDABProjectionArchitecture()->getXBlockNb();
	unsigned long long int yBlockNb = this->getCUDABProjectionArchitecture()->getYBlockNb();
	unsigned long long int zBlockNb = this->getCUDABProjectionArchitecture()->getZBlockNb();


	float taille_bloc_allocation,ratio_bloc_SDRAM;
	unsigned int N_phi_par_bloc;
	float taille_allocation,taille_allocation_sino,taille_allocation_vol,taille_SDRAM,ratio_allocation_SDRAM;
	taille_SDRAM=(float)prop_device.totalGlobalMem;
	taille_allocation_sino=sizeof(half)*uSinogramPixelNb*vSinogramPixelNb*projectionSinogramNb/gpuNb;//il faut allouer 1 sinogramme
	taille_allocation_vol=nstreams*sizeof(half)*(size_t)(xThreadNb*xBlockNb)*(size_t)(yThreadNb*yBlockNb)*16.0;
	taille_allocation=taille_allocation_sino+taille_allocation_vol;
	ratio_allocation_SDRAM=taille_allocation/taille_SDRAM;
	printf("allocation : %.2f Go  SDRAM : %.2f Go ratio :%.2f\n",taille_allocation/((1024.0*1024.0*1024.0)),taille_SDRAM/(1024.0*1024.0*1024.0),ratio_allocation_SDRAM);

	N_phi_par_bloc=projectionSinogramNb;
	while (N_phi_par_bloc>2048) //limitations des texture 2D par layer => pas plus de 2048 layers
	{
		nb_bloc_phi*=2;
		N_phi_par_bloc/=2;
	}

	printf("nb_blocs_phi %d\n",nb_bloc_phi);
	printf("nb_blocs_z_par_device %d ",nb_bloc_z_par_device);
	taille_bloc_allocation=taille_allocation_sino/(nb_bloc_phi*nb_bloc_z_par_device)+taille_allocation_vol;
	ratio_bloc_SDRAM=taille_bloc_allocation/taille_SDRAM;
	printf("allocation : %.2f Go  (sino %.2f Go vol %.2f Go)  SDRAM : %.2f Go ratio :%.2f\n",taille_bloc_allocation/((1024.0*1024.0*1024.0)),(taille_allocation_sino/(nb_bloc_phi*nb_bloc_z_par_device))/((1024.0*1024.0*1024.0)),taille_allocation_vol/((1024.0*1024.0*1024.0)),taille_SDRAM/(1024.0*1024.0*1024.0),ratio_bloc_SDRAM);

	while(taille_allocation_vol/taille_SDRAM>=0.5)
	{
		if (nstreams>1)
			nstreams/=2;

		taille_allocation_vol=nstreams*sizeof(half)*(size_t)(xThreadNb*xBlockNb)*(size_t)(yThreadNb*yBlockNb)*16.0;;
		taille_allocation=taille_allocation_sino+taille_allocation_vol;
		taille_bloc_allocation=taille_allocation_sino/(nb_bloc_phi*nb_bloc_z_par_device)+taille_allocation_vol;
		ratio_bloc_SDRAM=taille_bloc_allocation/taille_SDRAM;
	}

	while(ratio_bloc_SDRAM>=0.7)
	{
		nb_bloc_z_par_device*=2;
		printf("%d ",nb_bloc_z_par_device);
		taille_bloc_allocation=taille_allocation_sino/(nb_bloc_phi*nb_bloc_z_par_device)+taille_allocation_vol;
		ratio_bloc_SDRAM=taille_bloc_allocation/taille_SDRAM;
		printf("allocation : %.2f Go (sino %.2f Go vol %.2f Go) SDRAM : %.2f Go ratio :%.2f\n",taille_bloc_allocation/((1024.0*1024.0*1024.0)),(taille_allocation_sino/(nb_bloc_phi*nb_bloc_z_par_device))/((1024.0*1024.0*1024.0)),taille_allocation_vol/((1024.0*1024.0*1024.0)),taille_SDRAM/(1024.0*1024.0*1024.0),ratio_bloc_SDRAM);
	}

	unsigned int N_zn_par_carte;
	unsigned int N_zn_par_solverthread;
	unsigned int N_ligne_par_solverthread;
	unsigned int N_zn_par_kernel;
	unsigned int *num_bloc;
	num_bloc=(unsigned int*)malloc(sizeof(unsigned int)*nb_bloc_z_par_device*gpuNb);
	unsigned int *num_device;
	num_device=(unsigned int*)malloc(sizeof(unsigned int)*gpuNb);

	N_zn_par_carte=zVolumePixelNb/(gpuNb);
	N_zn_par_solverthread=N_zn_par_carte/(nb_bloc_z_par_device);
	N_zn_par_kernel=16;
	N_ligne_par_solverthread=(int)(N_zn_par_solverthread/N_zn_par_kernel);

	while (N_ligne_par_solverthread%nstreams!=0)
	{
		nstreams/=2;
	}

	this->getCUDABProjectionArchitecture()->setBProjectionStreamsNb(nstreams);

	printf("N_zn_par_carte %d N_zn_par_solverthread %d nb_bloc_z_par_device %d \n",N_zn_par_carte,N_zn_par_solverthread,nb_bloc_z_par_device);
	hipEvent_t *start_thread;
	hipEvent_t *stop_thread;

	start_thread=(hipEvent_t *)malloc(gpuNb*nb_bloc_z_par_device*sizeof(hipEvent_t));
	stop_thread=(hipEvent_t *)malloc(gpuNb*nb_bloc_z_par_device*sizeof(hipEvent_t));

	plan=(TGPUplan_retro_half<Volume_CPU_half, Sinogram3D_CPU_half>*)malloc(gpuNb*nb_bloc_z_par_device*sizeof(TGPUplan_retro_half<Volume_CPU_half, Sinogram3D_CPU_half>));
	threadID=(std::thread **)malloc(gpuNb*nb_bloc_z_par_device*sizeof(std::thread *));

	for(device=0;device<gpuNb;device++)
	{
		num_device[device]=device;

		//cout << "********** Start Constant Copy **********" << endl;
		//cout << "BackProjection Constant Copy on device n° " << device << endl;
		hipSetDevice(device);
		this->copyConstantGPU();
		//cout << "********** End BackProjection Constant Copy **********" << endl;

		for (int n=0;n<nb_bloc_z_par_device;n++){
			if (device%2==1)
				num_bloc[n+device*nb_bloc_z_par_device]=(nb_bloc_z_par_device-1)-n;
			else
				num_bloc[n+device*nb_bloc_z_par_device]=n;
			//printf("n %d device %d num_device %d n+device*nb_bloc_z_par_device %d num_bloc %d\n",n,device,num_device[device],n+device*nb_bloc_z_par_device,num_bloc[n+device*nb_bloc_z_par_device]);
			checkCudaErrors(hipEventCreate(start_thread+n+device*nb_bloc_z_par_device));
			checkCudaErrors(hipEventCreate(stop_thread+n+device*nb_bloc_z_par_device));
			plan[n+device*nb_bloc_z_par_device].device=device;
			plan[n+device*nb_bloc_z_par_device].fdk=this->getFdk();
			plan[n+device*nb_bloc_z_par_device].volume_h=this->getVolume();
			plan[n+device*nb_bloc_z_par_device].sinogram_h=sinogram;
			plan[n+device*nb_bloc_z_par_device].acquisition=this->getAcquisition();
			plan[n+device*nb_bloc_z_par_device].detector=this->getDetector();
			plan[n+device*nb_bloc_z_par_device].cudabackprojectionArchitecture=this->getCUDABProjectionArchitecture();
			plan[n+device*nb_bloc_z_par_device].N_zn_par_carte=N_zn_par_carte;
			plan[n+device*nb_bloc_z_par_device].N_zn_par_solverthread=N_zn_par_solverthread;
		}
	}

	if (nb_bloc_phi==1){
		for (int n=0;n<nb_bloc_z_par_device;n++){
			for(device=0;device<gpuNb;device++){
				int zn_start,zn_prime_start,zn_prime_stop,vn_start,vn_stop;
				float vn_prime_start,vn_prime_stop;

				zn_start=num_bloc[n+device*nb_bloc_z_par_device]*N_zn_par_solverthread+num_device[device]*N_zn_par_carte;
				zn_prime_start=zn_start-zVolumeCenterPixel;
				zn_prime_stop=zn_prime_start+N_zn_par_solverthread;
				if(zn_prime_start>=0)
					vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod+(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;
				else
					vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod-(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;

				if(zn_prime_stop>=0)
					vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod-(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;
				else
					vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod+(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;

				vn_stop=((int)vn_prime_stop+1)+vDetectorCenterPixel;

				if (vn_stop<0)
					vn_stop=0;
				if (vn_stop>=vDetectorPixelNb)
					vn_stop=vDetectorPixelNb-1;

				vn_start=((int)vn_prime_start-1)+vDetectorCenterPixel;

				if (vn_start<0)
					vn_start=0;

				//printf("device %d bloc %d zn_start %d zn_prime_start %d zn_prime_stop %d vn_prime_start %f vn_prime_stop %f vn_start %d vn_stop %d N_vn_par_solverthread %d\n",device,n,zn_start,zn_prime_start,zn_prime_stop,vn_prime_start,vn_prime_stop,vn_start,vn_stop,vn_stop-vn_start);

				plan[n+device*nb_bloc_z_par_device].zn_start=zn_start;
				plan[n+device*nb_bloc_z_par_device].vn_start=vn_start;
				plan[n+device*nb_bloc_z_par_device].N_vn_par_solverthread=(vn_stop-vn_start)+1;
				plan[n+device*nb_bloc_z_par_device].phi_start=0;
				plan[n+device*nb_bloc_z_par_device].N_phi_reduit=projectionSinogramNb;

				//threadID[device+n*nb_bloc_z_par_device] = cutStartThread((CUT_THREADROUTINE)solverThread, (void *)(plan + n+device*nb_bloc_z_par_device));
				threadID[device] = new std::thread((CUT_THREADROUTINE)solverThread, (void *)(plan + n+device*nb_bloc_z_par_device));

				//std::cout << "Joinable after construction:\n" << std::boolalpha;
				//std::cout << device << " : " << threadID[device]->joinable() << '\n';

			}
			//cutWaitForThreads(threadID+n*nb_bloc_z_par_device,gpuNb);
			for (int i = 0; i < gpuNb; i++)
			{
				if (threadID[i]->joinable()){
					//std::cout << i <<" joined:\n" << endl;
					threadID[i]->join();
				}
			}
		}


	}

	else
	{
		Volume_CPU_half* volume_temp_h = new Volume_CPU_half(this->getVolume()->getXVolumeSize(),this->getVolume()->getYVolumeSize(),this->getVolume()->getZVolumeSize(),this->getVolume()->getXVolumePixelNb(),this->getVolume()->getYVolumePixelNb(),this->getVolume()->getZVolumePixelNb(),NULL);

		for (int n=0;n<nb_bloc_z_par_device;n++){
			for(device=0;device<gpuNb;device++){
				int zn_start,zn_prime_start,zn_prime_stop,vn_start,vn_stop;
				float vn_prime_start,vn_prime_stop;

				zn_start=num_bloc[n+device*nb_bloc_z_par_device]*N_zn_par_solverthread+num_device[device]*N_zn_par_carte;
				zn_prime_start=zn_start-zVolumeCenterPixel;
				zn_prime_stop=zn_prime_start+N_zn_par_solverthread;
				if(zn_prime_start>=0)
					vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod+(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;
				else
					vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod-(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;

				if(zn_prime_stop>=0)
					vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod-(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;
				else
					vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod+(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;

				vn_stop=((int)vn_prime_stop+1)+vDetectorCenterPixel;

				if (vn_stop<0)
					vn_stop=0;
				if (vn_stop>=vDetectorPixelNb)
					vn_stop=vDetectorPixelNb-1;

				vn_start=((int)vn_prime_start-1)+vDetectorCenterPixel;

				if (vn_start<0)
					vn_start=0;

				//printf("device %d bloc %d zn_start %d zn_prime_start %d zn_prime_stop %d vn_prime_start %f vn_prime_stop %f vn_start %d vn_stop %d N_vn_par_solverthread %d\n",device,n,zn_start,zn_prime_start,zn_prime_stop,vn_prime_start,vn_prime_stop,vn_start,vn_stop,vn_stop-vn_start);

				plan[n+device*nb_bloc_z_par_device].zn_start=zn_start;
				plan[n+device*nb_bloc_z_par_device].vn_start=vn_start;
				plan[n+device*nb_bloc_z_par_device].N_vn_par_solverthread=(vn_stop-vn_start)+1;
				plan[n+device*nb_bloc_z_par_device].phi_start=0;
				plan[n+device*nb_bloc_z_par_device].N_phi_reduit=N_phi_par_bloc;
				threadID[device] = new std::thread((CUT_THREADROUTINE)solverThread, (void *)(plan + n+device*nb_bloc_z_par_device));

				//std::cout << "Joinable after construction:\n" << std::boolalpha;
				//std::cout << device << " : " << threadID[device]->joinable() << '\n';

			}
			//cutWaitForThreads(threadID+n*nb_bloc_z_par_device,gpuNb);
			for (int i = 0; i < gpuNb; i++)
			{
				if (threadID[i]->joinable()){
					//std::cout << i <<" joined:\n" << endl;
					threadID[i]->join();
				}
			}

		}


		for (int n=0;n<nb_bloc_z_par_device;n++){
			for(device=0;device<gpuNb;device++){
				int zn_start,zn_prime_start,zn_prime_stop,vn_start,vn_stop;
				float vn_prime_start,vn_prime_stop;

				zn_start=num_bloc[n+device*nb_bloc_z_par_device]*N_zn_par_solverthread+num_device[device]*N_zn_par_carte;
				zn_prime_start=zn_start-zVolumeCenterPixel;
				zn_prime_stop=zn_prime_start+N_zn_par_solverthread;
				if(zn_prime_start>=0)
					vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod+(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;
				else
					vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod-(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;

				if(zn_prime_stop>=0)
					vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod-(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;
				else
					vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod+(float)(xVolumePixelNb)*xVolumePixelSize/2.0))/vDetectorPixelSize;

				vn_stop=((int)vn_prime_stop+1)+vDetectorCenterPixel;

				if (vn_stop<0)
					vn_stop=0;
				if (vn_stop>=vDetectorPixelNb)
					vn_stop=vDetectorPixelNb-1;

				vn_start=((int)vn_prime_start-1)+vDetectorCenterPixel;

				if (vn_start<0)
					vn_start=0;

				//printf("device %d bloc %d zn_start %d zn_prime_start %d zn_prime_stop %d vn_prime_start %f vn_prime_stop %f vn_start %d vn_stop %d N_vn_par_solverthread %d\n",device,n,zn_start,zn_prime_start,zn_prime_stop,vn_prime_start,vn_prime_stop,vn_start,vn_stop,vn_stop-vn_start);

				plan[n+device*nb_bloc_z_par_device].volume_h=volume_temp_h;
				plan[n+device*nb_bloc_z_par_device].zn_start=zn_start;
				plan[n+device*nb_bloc_z_par_device].vn_start=vn_start;
				plan[n+device*nb_bloc_z_par_device].N_vn_par_solverthread=(vn_stop-vn_start)+1;
				plan[n+device*nb_bloc_z_par_device].phi_start=N_phi_par_bloc;
				plan[n+device*nb_bloc_z_par_device].N_phi_reduit=N_phi_par_bloc;
				threadID[device] = new std::thread((CUT_THREADROUTINE)solverThread, (void *)(plan + n+device*nb_bloc_z_par_device));

				//std::cout << "Joinable after construction:\n" << std::boolalpha;
				//std::cout << device << " : " << threadID[device]->joinable() << '\n';

			}
			//cutWaitForThreads(threadID+n*nb_bloc_z_par_device,gpuNb);
			for (int i = 0; i < gpuNb; i++)
			{
				if (threadID[i]->joinable()){
					//std::cout << i <<" joined:\n" << endl;
					threadID[i]->join();
				}
			}
		}

		printf("add on CPU bloc\n");
		this->getVolume()->addVolume(volume_temp_h);
		delete volume_temp_h;
	}


	for (int i = 0; i < gpuNb; i++)
	{

		delete threadID[i];

	}

	delete threadID;
	delete plan;
	delete start_thread;
	delete stop_thread;
	delete num_bloc;
	delete num_device;
}

/*void VIBackProjector_CPU_half::doBackProjectionSFTR(Volume_CPU_half *estimatedVolume,Sinogram3D_CPU_half* sinogram){}

void VIBackProjector_CPU_half::weightedCoeffDiagHTVHSFTR(Volume_CPU_half *coeffDiag,Sinogram3D_CPU_half *weights){}

void VIBackProjector_CPU_half::doBackProjectionSFTR_allCPU(Volume_CPU_half *estimatedVolume,Sinogram3D_CPU_half* sinogram){}*/

CUT_THREADPROC VIBackProjector_CPU_half::solverThread(TGPUplan_retro_half<Volume_CPU_half, Sinogram3D_CPU_half> *plan)
{


	int phi_start=0;
	size_t size_volume;
	unsigned long int kligne;
	int zn_start,zn_prime_start,zn_prime_stop,vn_start,vn_stop,vn_stop_old,N_ligne_par_carte;
	float N_zn_restant,vn_prime_start,vn_prime_stop;
	half* volume_d;

	hipChannelFormatDesc channelDesc;
	hipArray *sino_cu_3darray;
	hipMemcpy3DParms myparms_sino_3Darray = {0};

	unsigned int gpuNb = plan->cudabackprojectionArchitecture->getComputingUnitNb();
	hipStream_t *streams;
	int nstreams=plan->cudabackprojectionArchitecture->getBProjectionStreamsNb();

	float fdd = plan->acquisition->getFocusDetectorDistance();
	float fod = plan->acquisition->getFocusObjectDistance();
	float vDetectorPixelSize = plan->detector->getVDetectorPixelSize();
	float vDetectorPixelNb = plan->detector->getVDetectorPixelNb();
	float vDetectorCenterPixel = plan->detector->getVDetectorCenterPixel();

	float xVolumePixelSize = plan->volume_h->getXVolumePixelSize();
	float zVolumeCenterPixel = plan->volume_h->getZVolumeCenterPixel();
	unsigned long int xVolumePixelNb = plan->volume_h->getXVolumePixelNb();
	unsigned long int yVolumePixelNb = plan->volume_h->getYVolumePixelNb();
	unsigned long int zVolumePixelNb = plan->volume_h->getZVolumePixelNb();

	unsigned short xThreadNb = plan->cudabackprojectionArchitecture->getXThreadNb();
	unsigned short yThreadNb = plan->cudabackprojectionArchitecture->getYThreadNb();
	unsigned short zThreadNb = plan->cudabackprojectionArchitecture->getZThreadNb();
	unsigned short xBlockNb = plan->cudabackprojectionArchitecture->getXBlockNb();
	unsigned short yBlockNb = plan->cudabackprojectionArchitecture->getYBlockNb();
	unsigned short zBlockNb = plan->cudabackprojectionArchitecture->getZBlockNb();

	unsigned long int sinoU_h = plan->sinogram_h->getUSinogramPixelNb();
	unsigned long int sinoV_h = plan->sinogram_h->getVSinogramPixelNb();
	unsigned long int sinoPhi_h = plan->sinogram_h->getProjectionSinogramNb();

	half* dataSinogram = plan->sinogram_h->getDataSinogram();
	half* dataVolume = plan->volume_h->getVolumeData();

	//Set device
	checkCudaErrors(hipSetDevice(plan->device));

	hipEvent_t start_solverthread,stop_solverthread;
	checkCudaErrors(hipEventCreate(&start_solverthread));
	checkCudaErrors(hipEventCreate(&stop_solverthread));
	checkCudaErrors(hipEventRecord(start_solverthread, NULL));

	streams = (hipStream_t*)malloc((nstreams+1)*sizeof(hipStream_t));

	for(int i=0; i<nstreams+1 ; i++)
		checkCudaErrors(hipStreamCreate(&streams[i])) ;

	N_ligne_par_carte=(int)(plan->N_zn_par_solverthread/16);

	//printf("Streams Nb : %d\n",nstreams);

	hipEvent_t *event;
	event=(hipEvent_t *)malloc(nstreams*sizeof(hipEvent_t));
	for(int i=0;i<nstreams;i++)
		checkCudaErrors(hipEventCreate(event+i));

	N_zn_restant=((float)(plan->N_zn_par_solverthread)/16)-N_ligne_par_carte;

	if(N_zn_restant>0)
	{
		N_ligne_par_carte+=1;
	}
	printf("GPU%d N_zn_par_solverthread: %d N_ligne_par_carte:%d N_zn_restant; %f\n",plan->device,plan->N_zn_par_solverthread, N_ligne_par_carte,N_zn_restant);

	//Decoupage en thread
	dim3 dimBlock(xThreadNb,yThreadNb,zThreadNb);
	dim3 dimGrid(xBlockNb,yBlockNb,zBlockNb);
	size_volume=(size_t)(xThreadNb*xBlockNb)*(size_t)(yThreadNb*yBlockNb)*(size_t)16;
	checkCudaErrors(hipMalloc((void**) &(volume_d), sizeof(half)*size_volume*(size_t)nstreams));

	//Mise des sinogram en texture 2D layered
	channelDesc = hipCreateChannelDesc(sizeof(half)*8, 0, 0, 0, hipChannelFormatKindFloat);

	checkCudaErrors(hipMalloc3DArray(&sino_cu_3darray, &channelDesc, make_hipExtent((size_t)sinoU_h,(size_t)plan->N_vn_par_solverthread,(size_t)plan->N_phi_reduit), hipArrayLayered));

	myparms_sino_3Darray.kind = hipMemcpyHostToDevice;
	myparms_sino_3Darray.dstArray = sino_cu_3darray;


	sinogram_tex0.addressMode[0] = hipAddressModeBorder;
	sinogram_tex0.addressMode[1] = hipAddressModeBorder;
	sinogram_tex0.filterMode = hipFilterModeLinear;
	sinogram_tex0.normalized = false; // access with normalized texture coordinates

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(sinogram_tex0, sino_cu_3darray, channelDesc));

	phi_start=plan->phi_start;
	kligne=0;

	checkCudaErrors(hipMemset((void*) volume_d, 0,sizeof(half)*size_volume*nstreams));

	for(kligne=0;kligne<((N_ligne_par_carte/nstreams));kligne++)
	{
		vn_stop_old=vn_stop;

		int i = 0;

		zn_start=plan->zn_start+(i+kligne*nstreams)*16;
		zn_prime_start=zn_start-zVolumeCenterPixel;
		zn_prime_stop=zn_prime_start+16;
		if(zn_prime_stop>=0)
			vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod-(float)xVolumePixelNb*xVolumePixelSize/2.0))/vDetectorPixelSize;
		else
			vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod+(float)xVolumePixelNb*xVolumePixelSize/2.0))/vDetectorPixelSize;

		vn_stop=((int)vn_prime_stop+1)+vDetectorCenterPixel;

		if (vn_stop<0)
			vn_stop=0;
		if (vn_stop>=vDetectorPixelNb)
			vn_stop=vDetectorPixelNb-1;

		if(kligne!=0)
			vn_start=vn_stop_old+1;
		else{
			if(zn_prime_start>=0)
				vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod+(float)xVolumePixelNb*xVolumePixelSize/2.0))/vDetectorPixelSize;
			else
				vn_prime_start=((float)fdd*(float)zn_prime_start*xVolumePixelSize/((float)fod-(float)xVolumePixelNb*xVolumePixelSize/2.0))/vDetectorPixelSize;
			vn_start=((int)vn_prime_start-1)+vDetectorCenterPixel;

			if (vn_start<0)
				vn_start=0;
			if ((vn_start-plan->vn_start)<0)
				vn_start=plan->vn_start;
		}

		//printf("%d zn_start %d zn_prime_start %d zn_prime_stop %d vn_prime_start %f vn_prime_stop %f vn_start %d vn_stop %d(vn_stop-vn_start)+1 %d\n",i,zn_start,zn_prime_start,zn_prime_stop,vn_prime_start,vn_prime_stop,vn_start,vn_stop,(vn_stop-vn_start)+1);

		if (kligne!=0)
			hipStreamWaitEvent( streams[i], event[nstreams-1],0 );

		for (unsigned int phi=plan->phi_start;phi<plan->phi_start+plan->N_phi_reduit;phi++)
		{
			myparms_sino_3Darray.srcPos = make_hipPos(0,0,0);//make_hipPos(0,plan->vn_start,phi);
			myparms_sino_3Darray.dstPos = make_hipPos(0,vn_start-plan->vn_start,phi-plan->phi_start);
			myparms_sino_3Darray.srcPtr = make_hipPitchedPtr(dataSinogram+phi*sinoU_h*sinoV_h+vn_start*sinoU_h, sinoU_h*sizeof(half), sinoU_h, sinoV_h);
			myparms_sino_3Darray.extent = make_hipExtent(sinoU_h,(vn_stop-vn_start)+1,1);

			if (((vn_stop-vn_start)+1)>0)
				checkCudaErrors(hipMemcpy3DAsync(&myparms_sino_3Darray,streams[i]));
		}

		hipEventRecord (event[i], streams[i]);

		backprojection_VIB_kernel_v0_16reg_half<<< dimGrid, dimBlock,0,streams[i]>>>((unsigned short*)volume_d+size_volume*i,phi_start,zn_start,plan->N_phi_reduit,plan->vn_start);

		checkCudaErrors(hipMemcpyAsync(dataVolume+zn_start*xVolumePixelNb*yVolumePixelNb, volume_d+size_volume*i, size_volume*sizeof(half),hipMemcpyDeviceToHost,streams[i])) ;

		for(i=1; i < nstreams; i++)
		{
			vn_stop_old=vn_stop;
			zn_start=plan->zn_start+(i+kligne*nstreams)*16;

			zn_prime_start=zn_start-zVolumeCenterPixel;
			zn_prime_stop=zn_prime_start+16;

			if(zn_prime_stop>=0)
				vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod-(float)xVolumePixelNb*xVolumePixelSize/2.0))/vDetectorPixelSize;
			else
				vn_prime_stop=((float)fdd*(float)zn_prime_stop*xVolumePixelSize/((float)fod+(float)xVolumePixelNb*xVolumePixelSize/2.0))/vDetectorPixelSize;

			vn_stop=((int)vn_prime_stop+1)+vDetectorCenterPixel;

			if (vn_stop<0)
				vn_stop=0;
			if (vn_stop>=vDetectorPixelNb)
				vn_stop=vDetectorPixelNb-1;

			vn_start=vn_stop_old+1;

			//printf("%i zn_start %d zn_prime_start %d zn_prime_stop %d vn_prime_start %f vn_prime_stop %f vn_start %d vn_stop %d (vn_stop-vn_start)+1 %d\n",i,zn_start,zn_prime_start,zn_prime_stop,vn_prime_start,vn_prime_stop,vn_start,vn_stop,(vn_stop-vn_start)+1);


			if (((vn_stop-vn_start)+1)>0)
				hipStreamWaitEvent( streams[i], event[i-1],0 );



			for (unsigned int phi=plan->phi_start;phi<plan->phi_start+plan->N_phi_reduit;phi++){

				myparms_sino_3Darray.srcPos = make_hipPos(0,0,0);
				myparms_sino_3Darray.dstPos = make_hipPos(0,vn_start-plan->vn_start,phi-plan->phi_start);
				myparms_sino_3Darray.srcPtr = make_hipPitchedPtr(dataSinogram+phi*sinoU_h*sinoV_h+vn_start*sinoU_h, sinoU_h*sizeof(half), sinoU_h,sinoV_h);
				myparms_sino_3Darray.extent = make_hipExtent(sinoU_h,(vn_stop-vn_start)+1,1);

				if (((vn_stop-vn_start)+1)>0)
					checkCudaErrors(hipMemcpy3DAsync(&myparms_sino_3Darray,streams[i]));
			}

			hipEventRecord (event[i], streams[i]);

			backprojection_VIB_kernel_v0_16reg_half<<< dimGrid, dimBlock,0,streams[i]>>>((unsigned short*)volume_d+size_volume*i,phi_start,zn_start,plan->N_phi_reduit,plan->vn_start);

			checkCudaErrors(hipMemcpyAsync(dataVolume+zn_start*xVolumePixelNb*yVolumePixelNb, volume_d+size_volume*i, size_volume*sizeof(half),hipMemcpyDeviceToHost,streams[i])) ;
		}
	}
	checkCudaErrors(hipFreeArray(sino_cu_3darray));
	checkCudaErrors(hipFree(volume_d));

	for(int i=0 ; i < nstreams ; i++)
		checkCudaErrors(hipEventDestroy(event[i]));

	free(event);

	for(int i = 0 ; i < nstreams ; i++)
		checkCudaErrors(hipStreamDestroy(streams[i]));

	free(streams);

	checkCudaErrors(hipEventRecord(stop_solverthread, NULL));
	checkCudaErrors(hipEventSynchronize(stop_solverthread));

	hipEventDestroy(start_solverthread);
	hipEventDestroy(stop_solverthread);

}

/* VIBackProjector_GPU definition */
VIBackProjector_GPU_half::VIBackProjector_GPU_half(Acquisition* acquisition, Detector* detector, CUDABProjectionArchitecture* cudabackprojectionArchitecture,Volume_GPU_half* volume,char fdk) : BackProjector_half<Volume_GPU_half,Sinogram3D_GPU_half>(acquisition, detector, cudabackprojectionArchitecture,volume,fdk)
{
	//cout << "********** Start Constant Copy **********" << endl;
	//cout << "BackProjection Constant Copy on device n° " << 0 << endl;
	checkCudaErrors(hipSetDevice(0));
	this->copyConstantGPU();
	//cout << "********** End BackProjection Constant Copy **********" << endl;
																																																																												}

VIBackProjector_GPU_half::~VIBackProjector_GPU_half(){}

void VIBackProjector_GPU_half::doBackProjection(Volume_GPU_half* estimatedVolume,Sinogram3D_GPU_half* sinogram)
{
	std::cout << "VI BackProjection all on GPU" << std::endl;

	unsigned long int uSinogramPixelNb = sinogram->getUSinogramPixelNb();
	unsigned long int vSinogramPixelNb = sinogram->getVSinogramPixelNb();
	unsigned long int projectionSinogramPixelNb = sinogram->getProjectionSinogramNb();

	half* sinogramData = sinogram->getDataSinogram();

	hipChannelFormatDesc channelDesc;
	hipArray *sino_cu_3darray;
	hipMemcpy3DParms myparms_sino_3Darray = {0};

	this->setVolume(estimatedVolume);

	//Set device
	//CUDA_VISIBLE_DEVICES=0;
	checkCudaErrors(hipSetDevice(0));


	//Decoupage en thread
	dim3 dimBlock(this->getCUDABProjectionArchitecture()->getXThreadNb(),this->getCUDABProjectionArchitecture()->getYThreadNb(),this->getCUDABProjectionArchitecture()->getZThreadNb());
	dim3 dimGrid(this->getVolume()->getXVolumePixelNb()/this->getCUDABProjectionArchitecture()->getXThreadNb(), this->getVolume()->getYVolumePixelNb()/this->getCUDABProjectionArchitecture()->getYThreadNb(), this->getVolume()->getZVolumePixelNb()/(this->getCUDABProjectionArchitecture()->getZThreadNb()*16));

	//Mise des sinogram en texture 2D layered
	channelDesc = hipCreateChannelDesc(8*sizeof(half), 0, 0, 0, hipChannelFormatKindFloat);

	checkCudaErrors(hipMalloc3DArray(&sino_cu_3darray, &channelDesc, make_hipExtent(uSinogramPixelNb,vSinogramPixelNb,projectionSinogramPixelNb), hipArrayLayered));

	myparms_sino_3Darray.srcPos = make_hipPos(0,0,0);
	myparms_sino_3Darray.dstPos = make_hipPos(0,0,0);

	myparms_sino_3Darray.srcPtr = make_hipPitchedPtr(sinogramData, uSinogramPixelNb*sizeof(half), uSinogramPixelNb,vSinogramPixelNb);



	myparms_sino_3Darray.dstArray = sino_cu_3darray;
	myparms_sino_3Darray.extent = make_hipExtent(uSinogramPixelNb,vSinogramPixelNb,projectionSinogramPixelNb);
	myparms_sino_3Darray.kind = hipMemcpyDeviceToDevice;
	checkCudaErrors(hipMemcpy3D(&myparms_sino_3Darray));

	sinogram_tex0.addressMode[0] = hipAddressModeBorder;
	sinogram_tex0.addressMode[1] = hipAddressModeBorder;
	sinogram_tex0.filterMode = hipFilterModeLinear;
	sinogram_tex0.normalized = false;    // access with normalized texture coordinates

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(sinogram_tex0, sino_cu_3darray, channelDesc));

	backprojection_VIB_kernel_v0_16reg_half_UM<<< dimGrid, dimBlock>>>(this->getVolume()->getVolumeData(),projectionSinogramPixelNb);
	hipDeviceSynchronize();

	checkCudaErrors(hipFreeArray(sino_cu_3darray));
}

/*void VIBackProjector_GPU_half::doBackProjectionSFTR(Volume_GPU_half* estimatedVolume,Sinogram3D_GPU_half* sinogram){}

void VIBackProjector_GPU_half::weightedCoeffDiagHTVHSFTR(Volume_GPU_half* coeffDiag,Sinogram3D_GPU_half* weights){}

void VIBackProjector_GPU_half::doBackProjectionSFTR_allCPU(Volume_GPU_half* estimatedVolume,Sinogram3D_GPU_half* sinogram){}*/

#include "BackProjector_instances_CPU.cu"
#include "BackProjector_instances_GPU.cu"
